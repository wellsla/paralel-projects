/*
Welliton Slaviero
Matrícula: 178342

Meu dispositivo: {
    CPU: AMD Ryzen 5 5600X 6-Core Processor,
    Disco: KINGSTON SNV2S1000G, Capacidade: 932 GB,
    Memória: 16384 MB, Velocidade: 2400 MHz,
    GPU: NVIDIA GeForce RTX 3060 Ti,
    Informações CUDA do meu GPU: {
        Quantidade de devices: 1
        Device 0:
        Nome do device: NVIDIA GeForce RTX 3060 Ti
        Warp Size: 32
        Número máximo de threads por bloco: 1024
        Número máximo de threads por bloco por dimensão (X, Y, Z): (1024, 1024, 64)
        Número máximo de threads por grid por dimensão (X, Y, Z): (2147483647, 65535, 65535)
        Quantidade de multiprocessadores: 38
        Número máximo de threads por multiprocessador: 1536
    }
}

Implementação Mestre-Escravo com CUDA

RECURSOS DE PROGRAMAÇÃO PARALELA UTILIZADOS:
- Algoritmo: Mestre-Escravo adaptado para CUDA
- Mestre (CPU): criação e distribuição de tarefas, coleta e agregação de resultados
- Escravos (GPU Threads): processamento de tarefas individuais
- Comunicação: transferência estruturada de tarefas e resultados
- Balanceamento: divisão automática de carga entre workers
- Sincronização: barreira entre fases de distribuição e coleta

MOTIVO DA ESCOLHA:
Permite controle fino sobre distribuição de trabalho e facilita balanceamento de carga, seguindo o padrão Mestre-Escravo visto em aula.
*/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define MODULUS 2147483647
#define MULTIPLIER 48271
#define DEFAULT 123456789

static long seed = DEFAULT;

double Random(void) {
    const long Q = MODULUS / MULTIPLIER;
    const long R = MODULUS % MULTIPLIER;
    long t = MULTIPLIER * (seed % Q) - R * (seed / Q);
    seed = (t > 0 ? t : t + MODULUS);
    return ((double)seed / MODULUS);
}

typedef struct {
    double x, y, z, mass;
} Particle;

typedef struct {
    double xold, yold, zold;
    double fx, fy, fz;
} ParticleV;

__global__ void slaveKernel(Particle*, Particle*, ParticleV*, double*, int);
double masterSlaveComputeForces(Particle*, ParticleV*, int, float*, /*temporário*/ int);
void InitParticles(Particle[], ParticleV[], int);
double ComputeNewPos(Particle[], ParticleV[], int, double);

int main(int argc, char* argv[]) {
    Particle *particles;
    ParticleV *pv;
    int npart, cnt, /*temporário*/ ths;
    double sim_t;

    /*temporário*/ int cnt_main;
    if (argc >= 3) {// == 3) {
        // Via argumentos: ./nbody_cuda 16384 100
        npart = atoi(argv[1]);
        cnt = atoi(argv[2]);
        /*temporário*/ cnt_main = cnt;
        /*temporário*/ ths = atoi(argv[3]);
    } else {
        // Via redirecionamento: ./nbody_cuda < input.txt
        scanf("%d", &npart);
        scanf("%d", &cnt);
    }

    // Início da contagem de tempo total
    clock_t cpu_start = clock();
    float gpu_time = 0.0f;

    particles = (Particle *) malloc(sizeof(Particle)*npart);
    pv = (ParticleV *) malloc(sizeof(ParticleV)*npart);

    InitParticles(particles, pv, npart);
    sim_t = 0.0;

    hipSetDevice(0);

    while (cnt--) {
        double max_f;

        // USAR ABORDAGEM MESTRE-ESCRAVO
        max_f = masterSlaveComputeForces(particles, pv, npart, &gpu_time, /*temporário*/ ths);
        sim_t += ComputeNewPos(particles, pv, npart, max_f);
    }

    // Cálculo dos tempos
    clock_t cpu_end = clock();
    double total_time_ms = ((double)(cpu_end - cpu_start)) / CLOCKS_PER_SEC * 1000.0;
    double gpu_time_ms = gpu_time;
    double cpu_time_ms = total_time_ms - gpu_time_ms;

    /*temporário*/ // Tempos de execução
    fprintf(stdout, "Tempos de Execução (%d, %d, %d):\n", npart, cnt_main, ths);
    fprintf(stdout, "----------------------------------------\n");
    fprintf(stdout, "Tempo total: %.3f ms\n", total_time_ms);
    fprintf(stdout, "Tempo no kernel GPU: %.3f ms\n", gpu_time_ms);
    fprintf(stdout, "Tempo sequencial CPU: %.3f ms\n", cpu_time_ms);
    fprintf(stdout, "----------------------------------------\n");

    // APENAS coordenadas na saída padrão (para comparação)
    for (int i = 0; i < npart; i++) {
        fprintf(stdout, "%.5lf %.5lf %.5lf\n", particles[i].x, particles[i].y, particles[i].z);
    }

    free(particles);
    free(pv);
    return 0;
}

// KERNEL "ESCRAVO" - Cópia EXATA do ComputeForces original
__global__ void slaveKernel(Particle *myparticles, Particle *others, ParticleV *pv, double *forces, int npart) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= npart) return;

    // LÓGICA IDÊNTICA ao ComputeForces original
    int j;
    double xi, yi, rx, ry, mj, r, fx, fy, rmin;

    rmin = 100.0;
    xi = myparticles[i].x;
    yi = myparticles[i].y;
    fx = 0.0;
    fy = 0.0;

    for (j = 0; j < npart; j++) {
        rx = xi - others[j].x;
        ry = yi - others[j].y;
        mj = others[j].mass;
        r = rx * rx + ry * ry;
        /* ignore overlap and same particle */
        if (r == 0.0) continue;
        if (r < rmin) rmin = r;
        r = r * sqrt(r);
        fx -= mj * rx / r;
        fy -= mj * ry / r;
    }

    pv[i].fx += fx;
    pv[i].fy += fy;
    fx = sqrt(fx*fx + fy*fy)/rmin;
    forces[i] = fx;
}

// FUNÇÃO "MESTRE" - Versão simplificada que replica o comportamento original
double masterSlaveComputeForces(Particle *particles, ParticleV *pv, int npart, float *gpu_time, int ths) {
    static Particle *d_particles = NULL;
    static ParticleV *d_pv = NULL;
    static double *d_forces = NULL;
    static double *h_forces = NULL;
    static int allocated_npart = 0;
    static hipEvent_t start, stop;
    static int first_call = 1;

    if (first_call) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        first_call = 0;
    }

    // Inicialização (apenas uma vez ou quando tamanho muda)
    if (d_particles == NULL || allocated_npart != npart) {
        if (d_particles) {
            hipFree(d_particles);
            hipFree(d_pv);
            hipFree(d_forces);
            free(h_forces);
        }

        hipMalloc(&d_particles, npart * sizeof(Particle));
        hipMalloc(&d_pv, npart * sizeof(ParticleV));
        hipMalloc(&d_forces, npart * sizeof(double));
        h_forces = (double*)malloc(npart * sizeof(double));
        allocated_npart = npart;
    }

    // MESTRE: Enviar dados para "escravos" (GPU)
    hipMemcpy(d_particles, particles, npart * sizeof(Particle), hipMemcpyHostToDevice);
    hipMemcpy(d_pv, pv, npart * sizeof(ParticleV), hipMemcpyHostToDevice);

    // MESTRE: "Contratar" escravos para executar (1 escravo por partícula)
    int threadsPerBlock = /*temporário: a versão final será fixo como o professor pediu*/ ths; //256;
    int blocksPerGrid = (npart + threadsPerBlock - 1) / threadsPerBlock;

    // Medição GPU
    hipEventRecord(start);
    slaveKernel<<<blocksPerGrid, threadsPerBlock>>>(d_particles, d_particles, d_pv, d_forces, npart);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    *gpu_time += milliseconds;

    hipDeviceSynchronize();

    // MESTRE: Coletar resultados dos "escravos"
    hipMemcpy(pv, d_pv, npart * sizeof(ParticleV), hipMemcpyDeviceToHost);
    hipMemcpy(h_forces, d_forces, npart * sizeof(double), hipMemcpyDeviceToHost);

    // MESTRE: Encontrar força máxima (como no original)
    double max_f = 0.0;
    for (int i = 0; i < npart; i++) {
        if (h_forces[i] > max_f) max_f = h_forces[i];
    }

    return max_f;
}

void InitParticles(Particle particles[], ParticleV pv[], int npart) {
    for (int i = 0; i < npart; i++) {
        particles[i].x = Random();
        particles[i].y = Random();
        particles[i].z = Random();
        particles[i].mass = 1.0;
        pv[i].xold = particles[i].x;
        pv[i].yold = particles[i].y;
        pv[i].zold = particles[i].z;
        pv[i].fx = 0;
        pv[i].fy = 0;
        pv[i].fz = 0;
    }
}

double ComputeNewPos(Particle particles[], ParticleV pv[], int npart, double max_f) {
    int i;
    double a0, a1, a2;
    static double dt_old = 0.001, dt = 0.001;
    double dt_new;

    a0 = 2.0 / (dt * (dt + dt_old));
    a2 = 2.0 / (dt_old * (dt + dt_old));
    a1 = -(a0 + a2);

    for (i = 0; i < npart; i++) {
        double xi, yi;
        xi = particles[i].x;
        yi = particles[i].y;

        particles[i].x = (pv[i].fx - a1 * xi - a2 * pv[i].xold) / a0;
        particles[i].y = (pv[i].fy - a1 * yi - a2 * pv[i].yold) / a0;

        pv[i].xold = xi;
        pv[i].yold = yi;
        pv[i].fx = 0;
        pv[i].fy = 0;
    }

    dt_new = 1.0/sqrt(max_f);
    if (dt_new < 1.0e-6) dt_new = 1.0e-6;

    if (dt_new < dt) {
        dt_old = dt;
        dt = dt_new;
    } else if (dt_new > 4.0 * dt) {
        dt_old = dt;
        dt *= 2.0;
    }

    return dt_old;
}